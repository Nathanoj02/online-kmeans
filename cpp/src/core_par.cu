#include "hip/hip_runtime.h"
#include "core_par.hpp"

#include "error.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


namespace signals {

namespace cuda {

__global__
void k_means_kernel(
    uint8_t* d_img, uint8_t* d_assigned_img,
    uint8_t* d_prototypes, uint64_t *d_sums, uint64_t *d_counts,
    size_t img_height, size_t img_width, uint64_t k) 
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if core in image boundary
    if (i >= img_height || j >= img_width)
        return;
    
    uint8_t r = d_img[i * img_width * 3 + j * 3];
    uint8_t g = d_img[i * img_width * 3 + j * 3 + 1];
    uint8_t b = d_img[i * img_width * 3 + j * 3 + 2];

    float min_distance = MAXFLOAT;
    int assigned_prototype_index = -1;
    for (int p = 0; p < k; p++)
    {
        uint8_t prot_r = d_prototypes[p * 3];
        uint8_t prot_g = d_prototypes[p * 3 + 1];
        uint8_t prot_b = d_prototypes[p * 3 + 2];

        float distance_squared = (r - prot_r) * (r - prot_r) + (g - prot_g) * (g - prot_g) + (b - prot_b) * (b - prot_b);
        if (distance_squared < min_distance) {
            min_distance = distance_squared;
            assigned_prototype_index = p;
        }
    }
    d_assigned_img[i * img_width + j] = assigned_prototype_index;

    // Use atomic operations to safely update sums and counts
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3], r);
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3 + 1], g);
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3 + 2], b);
    atomicAdd((unsigned long long int*) &d_counts[assigned_prototype_index], 1);
}

void k_means(
    uint8_t* dst, uint8_t* img,
    size_t img_height, size_t img_width,
    uint64_t k, float_t stab_error)
{
    // Alloc space for CUDA arrays
    uint8_t *d_img, *d_assigned_img, *d_prototypes;
    uint64_t *d_sums, *d_counts; 
    SAFE_CALL( hipMalloc(&d_img, sizeof(uint8_t) * img_height * img_width * 3) );
    SAFE_CALL( hipMalloc(&d_assigned_img, sizeof(uint8_t) * img_height * img_width) );
    SAFE_CALL( hipMalloc(&d_prototypes, sizeof(uint8_t) * k * 3) );
    SAFE_CALL( hipMalloc(&d_sums, sizeof(uint64_t) * k * 3) );
    SAFE_CALL( hipMalloc(&d_counts, sizeof(uint64_t) * k) );
    
    // Copy data to CUDA (initial)
    SAFE_CALL( hipMemcpy(d_img, img, sizeof(uint8_t) * img_height * img_width * 3, hipMemcpyHostToDevice));

    srand((unsigned) time(NULL));

    // Create k prototypes with random values
    uint8_t* prototypes = (uint8_t*) malloc (sizeof(uint8_t) * k * 3);
    for (int i = 0; i < k * 3; i++) 
    {
        prototypes[i] = rand() % 256;
    }
    
    uint8_t* assigned_img = (uint8_t*) malloc (sizeof(uint8_t) * img_height * img_width);  // Map : pixels -> cluster number

    // Array for calculating means
    uint64_t* sums = (uint64_t*) malloc (sizeof(uint64_t) * k * 3);
    uint64_t* counts = (uint64_t*) malloc (sizeof(uint64_t) * k);

    uint8_t* old_prototypes = (uint8_t*) malloc (sizeof(uint8_t) * k * 3);

    // Zero array for copying into sums and counts
    uint64_t* zeros = (uint64_t*) calloc (k * 3, sizeof(uint64_t));
    
    DeviceInfo dev_info;
    find_best_grid(dev_info, img_height, img_width);

    // Square the error to match the distance
    stab_error = pow(stab_error, 2);
    bool bound_reached = false;

    // Loop until prototypes are stable
    for (int iteration_count = 0; !bound_reached; iteration_count++)
    {
        memcpy(old_prototypes, prototypes, k * 3 * sizeof(uint8_t));    // Save old values for calculating differences
        
        // Copy to CUDA
        SAFE_CALL( hipMemcpy(d_prototypes, old_prototypes, sizeof(uint8_t) * k * 3, hipMemcpyHostToDevice));
        SAFE_CALL( hipMemset(d_sums, 0, sizeof(uint64_t) * k * 3));
        SAFE_CALL( hipMemset(d_counts, 0, sizeof(uint64_t) * k));

        // Kernel call
        dim3 dim_grid = dim3(dev_info.grid.x, dev_info.grid.y, dev_info.grid.z);
        dim3 dim_block = dim3(dev_info.block.x, dev_info.block.y, dev_info.block.z);
        k_means_kernel <<<dim_grid, dim_block>>> (
            d_img, d_assigned_img, d_prototypes, d_sums, d_counts,
            img_height, img_width, k
        );
        CHECK_CUDA_ERROR;

        // Copy data back to CPU
        SAFE_CALL( hipMemcpy(assigned_img, d_assigned_img, sizeof(uint8_t) * img_height * img_width, hipMemcpyDeviceToHost ));
        SAFE_CALL( hipMemcpy(sums, d_sums, sizeof(uint64_t) * k * 3, hipMemcpyDeviceToHost ));
        SAFE_CALL( hipMemcpy(counts, d_counts, sizeof(uint64_t) * k, hipMemcpyDeviceToHost ));

        // Update values of the prototypes to the means of the associated pixels
        for (int i = 0; i < k; i++)
        {
            if (counts[i] != 0)
            {
                prototypes[i * 3] = sums[i * 3] / counts[i];
                prototypes[i * 3 + 1] = sums[i * 3 + 1] / counts[i];
                prototypes[i * 3 + 2] = sums[i * 3 + 2] / counts[i];
            }
        }

        // Calculate differences
        bound_reached = true;

        for (int i = 0; i < k; i++)
        {
            uint8_t prot_r = prototypes[i * 3];
            uint8_t prot_g = prototypes[i * 3 + 1];
            uint8_t prot_b = prototypes[i * 3 + 2];
            uint8_t old_r = old_prototypes[i * 3];
            uint8_t old_g = old_prototypes[i * 3 + 1];
            uint8_t old_b = old_prototypes[i * 3 + 2];

            float distance_squared = pow(prot_r - old_r, 2) + pow(prot_g - old_g, 2) + pow(prot_b - old_b, 2);

            if (distance_squared > stab_error)
            {
                bound_reached = false;
                break;
            }
        }
    }

    // Substitute each pixel with the corresponding prototype value
    for (int i = 0; i < img_height; i++)
    {
        for (int j = 0; j < img_width; j++)
        {
            int index = assigned_img[i * img_width + j];
            dst[i * img_width * 3 + j * 3] = prototypes[index * 3];
            dst[i * img_width * 3 + j * 3 + 1] = prototypes[index * 3 + 1];
            dst[i * img_width * 3 + j * 3 + 2] = prototypes[index * 3 + 2];
        }
    }
}

static hipDeviceProp_t find_best_gpu()
{
    // Save properties of GPUs
    int dev_count;
    SAFE_CALL( hipGetDeviceCount(&dev_count) );

    // Save best GPU
    hipDeviceProp_t dev_prop;
    hipDeviceProp_t best_device_prop;

    for (int i = 0; i < dev_count; i++)
    {
        SAFE_CALL( hipGetDeviceProperties(&dev_prop, i) );
        
        if (i == 0 
            || (dev_prop.maxThreadsPerMultiProcessor * dev_prop.multiProcessorCount > 
                best_device_prop.maxThreadsPerMultiProcessor * best_device_prop.multiProcessorCount))
        {
            best_device_prop = dev_prop;
        }
    }
    return best_device_prop;
}

DeviceInfo& find_best_grid(DeviceInfo& device_info, std::size_t height, std::size_t width)
{
    auto best_device = find_best_gpu();

    std::size_t threadsPerBlockTemp;

    if(height * width > best_device.maxThreadsPerBlock / 2)
        threadsPerBlockTemp = best_device.maxThreadsPerBlock;
    else
        threadsPerBlockTemp = pow(2, ceil(log2(height * width)));
    
    std::size_t threadsPerBlockRow = pow(2, ceil(log2(sqrt(threadsPerBlockTemp))));

    device_info.block = {
        threadsPerBlockRow, 
        threadsPerBlockRow, 
        1,
    };
    device_info.grid = {
        width / threadsPerBlockRow, 
        height / threadsPerBlockRow,
        1
    };
    if (width % threadsPerBlockRow) device_info.grid.x++;
    if (height % threadsPerBlockRow) device_info.grid.y++;
    return device_info;
}



} // namespace cuda

} // namespace signals