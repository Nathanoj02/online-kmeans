#include "hip/hip_runtime.h"
#include "core_par.hpp"

#include "error.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>


namespace signals {

namespace cuda {

__global__
void k_means_kernel(
    uint8_t* d_img, uint8_t* d_assigned_img,
    uint8_t* d_prototypes, uint64_t *d_sums, uint64_t *d_counts,
    size_t img_height, size_t img_width, uint64_t k) 
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Check if core in image boundary
    if (i >= img_height || j >= img_width)
        return;
    
    uint8_t r = d_img[i * img_width * 3 + j * 3];
    uint8_t g = d_img[i * img_width * 3 + j * 3 + 1];
    uint8_t b = d_img[i * img_width * 3 + j * 3 + 2];

    float min_distance = MAXFLOAT;
    int assigned_prototype_index = -1;
    for (int p = 0; p < k; p++)
    {
        uint8_t prot_r = d_prototypes[p * 3];
        uint8_t prot_g = d_prototypes[p * 3 + 1];
        uint8_t prot_b = d_prototypes[p * 3 + 2];

        float distance_squared = (r - prot_r) * (r - prot_r) + (g - prot_g) * (g - prot_g) + (b - prot_b) * (b - prot_b);
        if (distance_squared < min_distance) {
            min_distance = distance_squared;
            assigned_prototype_index = p;
        }
    }
    d_assigned_img[i * img_width + j] = assigned_prototype_index;

    // Use atomic operations to safely update sums and counts
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3], r);
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3 + 1], g);
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3 + 2], b);
    atomicAdd((unsigned long long int*) &d_counts[assigned_prototype_index], 1);
}


__global__
void k_means_kernel_shared(
    uint8_t* d_img, uint8_t* d_assigned_img,
    uint8_t* d_prototypes, uint64_t *d_sums, uint64_t *d_counts,
    size_t img_height, size_t img_width, uint64_t k)
{
    extern __shared__ uint8_t prot_shared[];

    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Collaborative loading of prototypes into the shared memory
    int thread_idx = threadIdx.y * blockDim.x + threadIdx.x;
    if (thread_idx < k * 3) {
        prot_shared[thread_idx] = d_prototypes[thread_idx];
    }
    __syncthreads();

    // Check if core in image boundary
    if (i >= img_height || j >= img_width)
        return;

    // Algorithm as normal but with shared prototypes
    uint8_t r = d_img[i * img_width * 3 + j * 3];
    uint8_t g = d_img[i * img_width * 3 + j * 3 + 1];
    uint8_t b = d_img[i * img_width * 3 + j * 3 + 2];

    float min_distance = MAXFLOAT;
    int assigned_prototype_index = -1;
    for (int p = 0; p < k; p++)
    {
        uint8_t prot_r = prot_shared[p * 3];
        uint8_t prot_g = prot_shared[p * 3 + 1];
        uint8_t prot_b = prot_shared[p * 3 + 2];

        float distance_squared = (r - prot_r) * (r - prot_r) + (g - prot_g) * (g - prot_g) + (b - prot_b) * (b - prot_b);
        if (distance_squared < min_distance) {
            min_distance = distance_squared;
            assigned_prototype_index = p;
        }
    }
    d_assigned_img[i * img_width + j] = assigned_prototype_index;

    // Use atomic operations to safely update sums and counts
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3], r);
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3 + 1], g);
    atomicAdd((unsigned long long int*) &d_sums[assigned_prototype_index * 3 + 2], b);
    atomicAdd((unsigned long long int*) &d_counts[assigned_prototype_index], 1);
}


void k_means(
    uint8_t* dst, uint8_t* img,
    size_t img_height, size_t img_width,
    uint64_t k, float_t stab_error,
    const KmeansInfo& device_info, bool use_shared_mem)
{   
    // Copy data to CUDA (initial)
    SAFE_CALL( hipMemcpy(device_info.d_img, img, sizeof(uint8_t) * img_height * img_width * 3, hipMemcpyHostToDevice));

    srand((unsigned) time(NULL));

    // Create k prototypes with random values
    uint8_t* prototypes = (uint8_t*) malloc (sizeof(uint8_t) * k * 3);
    for (int i = 0; i < k * 3; i++) 
    {
        prototypes[i] = rand() % 256;
    }
    
    uint8_t* assigned_img = (uint8_t*) malloc (sizeof(uint8_t) * img_height * img_width);  // Map : pixels -> cluster number

    // Array for calculating means
    uint64_t* sums = (uint64_t*) malloc (sizeof(uint64_t) * k * 3);
    uint64_t* counts = (uint64_t*) malloc (sizeof(uint64_t) * k);

    uint8_t* old_prototypes = (uint8_t*) malloc (sizeof(uint8_t) * k * 3);

    // Zero array for copying into sums and counts
    uint64_t* zeros = (uint64_t*) calloc (k * 3, sizeof(uint64_t));
    
    DeviceInfo dev_info;
    find_best_grid(dev_info, img_height, img_width);

    bool bound_reached = false;

    // Loop until prototypes are stable
    for (int iteration_count = 0; !bound_reached; iteration_count++)
    {
        memcpy(old_prototypes, prototypes, k * 3 * sizeof(uint8_t));    // Save old values for calculating differences
        
        // Copy to CUDA
        SAFE_CALL( hipMemcpy(device_info.d_prototypes, old_prototypes, sizeof(uint8_t) * k * 3, hipMemcpyHostToDevice));
        SAFE_CALL( hipMemset(device_info.d_sums, 0, sizeof(uint64_t) * k * 3));
        SAFE_CALL( hipMemset(device_info.d_counts, 0, sizeof(uint64_t) * k));

        // Kernel call
        dim3 dim_grid = dim3(dev_info.grid.x, dev_info.grid.y, dev_info.grid.z);
        dim3 dim_block = dim3(dev_info.block.x, dev_info.block.y, dev_info.block.z);

        if (use_shared_mem) {
            k_means_kernel_shared <<<dim_grid, dim_block, k * 3 * sizeof(uint8_t)>>> (
                device_info.d_img, device_info.d_assigned_img, device_info.d_prototypes, 
                device_info.d_sums, device_info.d_counts,
                img_height, img_width, k
            );
        }
        else {
            k_means_kernel <<<dim_grid, dim_block>>> (
                device_info.d_img, device_info.d_assigned_img, device_info.d_prototypes, 
                device_info.d_sums, device_info.d_counts,
                img_height, img_width, k
            );
        }
        CHECK_CUDA_ERROR;

        // Copy data back to CPU
        SAFE_CALL( hipMemcpy(assigned_img, device_info.d_assigned_img, sizeof(uint8_t) * img_height * img_width, hipMemcpyDeviceToHost ));
        SAFE_CALL( hipMemcpy(sums, device_info.d_sums, sizeof(uint64_t) * k * 3, hipMemcpyDeviceToHost ));
        SAFE_CALL( hipMemcpy(counts, device_info.d_counts, sizeof(uint64_t) * k, hipMemcpyDeviceToHost ));

        // Update values of the prototypes to the means of the associated pixels
        for (int i = 0; i < k; i++)
        {
            if (counts[i] != 0)
            {
                prototypes[i * 3] = sums[i * 3] / counts[i];
                prototypes[i * 3 + 1] = sums[i * 3 + 1] / counts[i];
                prototypes[i * 3 + 2] = sums[i * 3 + 2] / counts[i];
            }
        }

        // Calculate differences
        bound_reached = true;

        for (int i = 0; i < k; i++)
        {
            uint8_t prot_r = prototypes[i * 3];
            uint8_t prot_g = prototypes[i * 3 + 1];
            uint8_t prot_b = prototypes[i * 3 + 2];
            uint8_t old_r = old_prototypes[i * 3];
            uint8_t old_g = old_prototypes[i * 3 + 1];
            uint8_t old_b = old_prototypes[i * 3 + 2];

            float distance_squared = pow(prot_r - old_r, 2) + pow(prot_g - old_g, 2) + pow(prot_b - old_b, 2);

            if (distance_squared > stab_error)
            {
                bound_reached = false;
                break;
            }
        }
    }

    // Substitute each pixel with the corresponding prototype value
    for (int i = 0; i < img_height; i++)
    {
        for (int j = 0; j < img_width; j++)
        {
            int index = assigned_img[i * img_width + j];
            dst[i * img_width * 3 + j * 3] = prototypes[index * 3];
            dst[i * img_width * 3 + j * 3 + 1] = prototypes[index * 3 + 1];
            dst[i * img_width * 3 + j * 3 + 2] = prototypes[index * 3 + 2];
        }
    }
}


KmeansInfo init_k_means(size_t img_height, size_t img_width, uint64_t k)
{
    KmeansInfo device_info;
    SAFE_CALL( hipMalloc(&device_info.d_img, sizeof(uint8_t) * img_height * img_width * 3) );
    SAFE_CALL( hipMalloc(&device_info.d_assigned_img, sizeof(uint8_t) * img_height * img_width) );
    SAFE_CALL( hipMalloc(&device_info.d_prototypes, sizeof(uint8_t) * k * 3) );
    SAFE_CALL( hipMalloc(&device_info.d_sums, sizeof(uint64_t) * k * 3) );
    SAFE_CALL( hipMalloc(&device_info.d_counts, sizeof(uint64_t) * k) );
    find_best_grid(device_info.dim, img_height, img_width);
    return device_info;
}


void deinit_k_means(KmeansInfo& device_info)
{
    SAFE_CALL( hipFree( device_info.d_img ) );
    SAFE_CALL( hipFree( device_info.d_assigned_img ) );
    SAFE_CALL( hipFree( device_info.d_prototypes ) );
    SAFE_CALL( hipFree( device_info.d_sums ) );
    SAFE_CALL( hipFree( device_info.d_counts ) );
}


static hipDeviceProp_t find_best_gpu()
{
    // Save properties of GPUs
    int dev_count;
    SAFE_CALL( hipGetDeviceCount(&dev_count) );

    // Save best GPU
    hipDeviceProp_t dev_prop;
    hipDeviceProp_t best_device_prop;

    for (int i = 0; i < dev_count; i++)
    {
        SAFE_CALL( hipGetDeviceProperties(&dev_prop, i) );
        
        if (i == 0 
            || (dev_prop.maxThreadsPerMultiProcessor * dev_prop.multiProcessorCount > 
                best_device_prop.maxThreadsPerMultiProcessor * best_device_prop.multiProcessorCount))
        {
            best_device_prop = dev_prop;
        }
    }
    return best_device_prop;
}


DeviceInfo& find_best_grid(DeviceInfo& device_info, std::size_t height, std::size_t width)
{
    auto best_device = find_best_gpu();

    std::size_t threadsPerBlockTemp;

    if(height * width > best_device.maxThreadsPerBlock / 2)
        threadsPerBlockTemp = best_device.maxThreadsPerBlock;
    else
        threadsPerBlockTemp = pow(2, ceil(log2(height * width)));
    
    std::size_t threadsPerBlockRow = pow(2, ceil(log2(sqrt(threadsPerBlockTemp))));

    device_info.block = {
        threadsPerBlockRow, 
        threadsPerBlockRow, 
        1,
    };
    device_info.grid = {
        width / threadsPerBlockRow, 
        height / threadsPerBlockRow,
        1
    };
    if (width % threadsPerBlockRow) device_info.grid.x++;
    if (height % threadsPerBlockRow) device_info.grid.y++;
    return device_info;
}



} // namespace cuda

} // namespace signals